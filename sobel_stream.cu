#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/core.hpp"
#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/imgcodecs.hpp"
#include <iostream>
#include "sobel.cuh"

using namespace cv;
using namespace std;

int main(int argc, char* argv[]) {
	
	int bdx = atoi(argv[1]);
	int bdy = atoi(argv[2]);

	hipError_t err;

	// int dev;
	// hipDeviceProp_t prop;
	// hipGetDevice(&dev);
	// hipGetDeviceProperties(&prop, dev);
	// cout << prop.sharedMemPerBlock << endl;

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	// hipEvent_t event;
	// hipEventCreate(&event);

	Mat image, norm_image;
    image = imread("license.jpg", 0);
    if(image.empty())               
    {
        cout <<  "Could not open or find the image" << endl;
        return -1;
    }
    
    normalize(image, norm_image, 0, 1, NORM_MINMAX, CV_32F);
    float *img = norm_image.ptr<float>(0);

	// cout << image.rows << " " << image.cols << endl;

    float maskx[9] = {-1,-2,-1,0,0,0,1,2,1};
    float masky[9] = {-1,0,1,-2,0,2,-1,0,1};

    float maskx1[3] = {1, 0, -1};
    float maskx2[3] = {1, 2, 1};

    float masky1[3] = {1, 2, 1};
    float masky2[3] = {1, 0, -1};

    float *dimg, *doutx, *douty, *doutput;
    err = hipMalloc((void **)&dimg, image.rows * image.cols * sizeof(float));
	err = hipMalloc((void **)&doutx, image.rows * image.cols * sizeof(float));
  	err = hipMalloc((void **)&douty, image.rows * image.cols * sizeof(float));
  	err = hipMalloc((void **)&doutput, image.rows * image.cols * sizeof(float));

  	float *himg, *houtx, *houty, *houtput;
  	err = hipHostAlloc((void **)&himg, image.rows*image.cols*sizeof(float), hipHostMallocDefault);
  	err = hipHostAlloc((void **)&houtx, image.rows*image.cols*sizeof(float), hipHostMallocDefault);
  	err = hipHostAlloc((void **)&houty, image.rows*image.cols*sizeof(float), hipHostMallocDefault);
  	err = hipHostAlloc((void **)&houtput, image.rows*image.cols*sizeof(float), hipHostMallocDefault);

  	memcpy(himg, norm_image.ptr<float>(), image.rows * image.cols * sizeof(float));
  	
  	float *dmaskx, *dmasky, *dmaskx1, *dmaskx2; 			// masky1 = maskx2 and masky2 = maskx1
  	err = hipMalloc((void **)&dmaskx, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmasky, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx1, 3 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx2, 3 * sizeof(float));
	
  	float *hmaskx, *hmasky, *hmaskx1, *hmaskx2;
  	hipHostAlloc((void **)&hmaskx, 9*sizeof(float), hipHostMallocDefault);
  	hipHostAlloc((void **)&hmasky, 9*sizeof(float), hipHostMallocDefault);
  	hipHostAlloc((void **)&hmaskx1, 3*sizeof(float), hipHostMallocDefault);
  	hipHostAlloc((void **)&hmaskx2, 3*sizeof(float), hipHostMallocDefault);

  	memcpy(hmaskx, maskx, 9 * sizeof(float));
  	memcpy(hmasky, masky, 9 * sizeof(float));
  	memcpy(hmaskx1, maskx1, 3 * sizeof(float));
  	memcpy(hmaskx2, maskx2, 3 * sizeof(float));

  	dim3 block(bdx, bdy);
	dim3 grid((image.cols + block.x - 1) / block.x, (image.rows + block.y - 1) / block.y);

	float *temp, *temp1;
	err = hipMalloc((void **)&temp, image.rows * image.cols * sizeof(float));
	err = hipMalloc((void **)&temp1, image.rows * image.cols * sizeof(float));
  	
  	hipEventRecord(start);

  	err = hipMemcpyAsync(dimg, himg, image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice, stream0);

  	
  	err = hipMemcpyAsync(dmaskx1, hmaskx1, 3 * sizeof(float), hipMemcpyHostToDevice, stream0);
  	
  	conv_kernel_horiz<<<grid, block, sizeof(float) * bdy * (bdx+2) + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream0>>>(dimg, dmaskx1, temp, image.rows, image.cols);
  	err = hipMemcpyAsync(dmaskx2, hmaskx2, 3 * sizeof(float), hipMemcpyHostToDevice, stream1);
  	
  	err = hipDeviceSynchronize();

  	conv_kernel_horiz<<<grid, block, sizeof(float) * bdy * (bdx+2) + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream0>>>(dimg, dmaskx2, temp1, image.rows, image.cols);
  	conv_kernel_vert<<<grid, block, sizeof(float) * (bdy+2) * bdx + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream1>>>(temp, dmaskx2, doutx, image.rows, image.cols);

  	conv_kernel_vert<<<grid, block, sizeof(float) * (bdy+2) * bdx + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream0>>>(temp1, dmaskx1, douty, image.rows, image.cols);
  	err = hipMemcpyAsync(houtx, doutx, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost, stream1);
  	
  	err = hipDeviceSynchronize();

  	magnitude<<<grid, block, 0, stream0>>>(doutx, douty, doutput, image.rows, image.cols);
   	err = hipMemcpyAsync(houty, douty, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost, stream1);

   	err = hipMemcpy(houtput, doutput, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost);
   	
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);

  	Mat out = Mat(image.rows, image.cols, CV_32F, houtput);
    Mat norm_out;
    normalize(out, norm_out, 0, 1, NORM_MINMAX, CV_32F);

  	cout << ms << endl;

  	Mat write_out;
    normalize(norm_out, write_out, 0, 255, NORM_MINMAX, CV_8U);
    imwrite("sobel1_stream.png", write_out);

  	err = hipFree(dimg);
  	err = hipFree(doutx);
  	err = hipFree(douty);
  	err = hipFree(doutput);
  	err = hipFree(dmaskx);
  	err = hipFree(dmasky);
  	err = hipFree(dmaskx1);
  	err = hipFree(dmaskx2);
  	err = hipFree(temp);
  	err = hipFree(temp1);

  	err = hipHostFree(himg);
  	err = hipHostFree(houtx);
  	err = hipHostFree(houty);
  	err = hipHostFree(houtput);
  	err = hipHostFree(hmaskx);
  	err = hipHostFree(hmasky);
  	err = hipHostFree(hmaskx1);
  	err = hipHostFree(hmaskx2);
  	
  	
  	cout << "Done!\n";
  	
  	return 0;
}
