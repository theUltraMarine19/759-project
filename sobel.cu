#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <iostream>
#include "sobel.cuh"
using namespace std;

__global__ void conv_kernel(const float* image, const float* mask, float* output, unsigned int r, unsigned int c) {
	
	int tidx = threadIdx.x, tidy = threadIdx.y;
	int bidx = blockIdx.x, bidy = blockIdx.y;
	int bdy = blockDim.y, bdx = blockDim.x;
	float avg_intensity = 0.5;

	// printf("%d %d %d %d\n", tidx, tidy, bidx, bidy);
	// if (tidx == 0 && tidy == 0 && bidx == 0 && bidy == 0) {
		
		// printf("%d %d\n", bidx, bidy);
		// for (int i = 0; i < 9; i++)
		// 	printf("%f ", mask[i]);
		// printf("\n");
	// }
	

	extern __shared__ float arr[];
	float* img = &arr[0]; 									
	float* msk = &arr[(bdx + 2) * (bdy + 2)]; 
	float* out = &arr[(bdx + 2) * (bdy + 2) + 3*3]; 		 

	long x_idx = tidx + (long)bdx * (long)bidx; 		// long since can be > 2^31 -1
	long y_idx = tidy + (long)bdy * (long)bidy;
	
	// load image elements in-lace
	if (x_idx < c && y_idx < r)
		img[(tidy+1)*(bdx+2) + tidx+1] = image[y_idx * c + x_idx];
	else
		img[(tidy+1)*(bdx+2) + tidx+1] = avg_intensity;

	
	if (tidx < 3 && tidy < 3)
		msk[tidy*3 + tidx] = mask[tidy*3 + tidx];

	
	if (tidx == 0 && tidy == 0) { // leftmost top corner
		
		if (x_idx >= 1 && y_idx >= 1)
			img[tidy*(bdx+2) + tidx] = image[(y_idx-1) * c + x_idx-1];
		else
			img[tidy*(bdx+2) + tidx] = avg_intensity;

	}
	else if (tidx == 0 && tidy == bdy - 1) { // leftmost bottom corner
		
		if (x_idx >= 1 && y_idx < r-1)
			img[(tidy+2)*(bdx+2) + tidx] = image[(y_idx+1) * c + x_idx-1];
		else
			img[(tidy+2)*(bdx+2) + tidx] = avg_intensity;		
	
	}
	else if (tidx == bdx - 1 && tidy == 0) { // rightmost top corner
		
		if (x_idx < c -1 && y_idx >= 1)
			img[tidy*(bdx+2) + tidx+2] = image[(y_idx-1) * c + x_idx+1];
		else
			img[tidy*(bdx+2) + tidx+2] = avg_intensity;

	}
	else if (tidx == bdx - 1 && tidy == bdy -1) { // rightmost bottom corner
		
		if (x_idx < c -1 && y_idx < r-1)
			img[(tidy+2)*(bdx+2) + tidx+2] = image[(y_idx+1) * c + x_idx+1];
		else
			img[(tidy+2)*(bdx+2) + tidx+2] = avg_intensity;
	
	}


	if (tidx == 0) { // leftmost col
		
		if (x_idx >= 1)
			img[(tidy+1)*(bdx+2) + tidx] = image[y_idx*c + x_idx-1];
		else
			img[(tidy+1)*(bdx+2) + tidx] = avg_intensity;
	
	}
	else if (tidx == bdx - 1) { // rightmost col
		
		if (x_idx < c-1)
			img[(tidy+1)*(bdx+2) + tidx+2] = image[y_idx*c + x_idx+1];
		else
			img[(tidy+1)*(bdx+2) + tidx+2] = avg_intensity;
	
	}
	
	if (tidy == 0) { // top row
		
		if (y_idx >= 1)
			img[tidy*(bdx+2) + tidx+1] = image[(y_idx-1)*c + x_idx];
		else
			img[tidy*(bdx+2) + tidx+1] = avg_intensity;
	
	}
	else if (tidy == bdy - 1) { // bottom row
	
		if (y_idx < r-1)
			img[(tidy+2)*(bdx+2) + tidx+1] = image[(y_idx+1)*c + x_idx];
		else
			img[(tidy+2)*(bdx+2) + tidx+1] = avg_intensity;
	
	}

	__syncthreads();

	if (tidx == 2 && tidy == 1 && bidx == 21 && bidy == 30) {
		
		for (int i = 0; i < bdy+2; i++) {
			for (int j = 0; j < bdx+2; j++ ) {
				printf("%f ", img[i*(bdx+2)+j]);	
			}
			printf("\n");	
		}

		for (int i = -1; i < bdy+1; i++) {
			for (int j = -1; j < bdx+1; j++ ) {
				printf("%f ", image[(bidy*bdy+i)*c+(bidx*bdx+j)]);	
			}
			printf("\n");	
		}		
	}

	out[tidy*bdx+tidx] = 0;
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			out[tidy*bdx+tidx] += img[(tidy+i)*(bdx+2) + (tidx+j)] * msk[i*3+j];	
		}		
	}

	__syncthreads();

	if (x_idx < c && y_idx < r)
		output[y_idx*c+x_idx] = out[tidy*bdy+tidx];

}

__host__ void conv(const float* image, const float* mask, float* output, unsigned int r, unsigned int c, unsigned int bdx, unsigned int bdy) {

	dim3 block(bdx, bdy);
	dim3 grid((c + block.x - 1) / block.x, (r + block.y - 1) / block.y);
	// cout << bdx << " " << bdy << " " << (c + block.x - 1) / block.x << " " << (r + block.y - 1) / block.y << endl;
	// for (int i = 0; i < r; i++) {
	// 	for (int j = 0; j < c; j++) {
	// 		cout << image[i*c+j] << " ";
	// 	}
	// 	cout << endl;
	// }

	conv_kernel<<<grid, block, sizeof(float) * (bdx + 2) * (bdy + 2) + 3 * 3 * sizeof(float) + sizeof(float) * bdx * bdy>>>(image, mask, output, r, c);
	
	hipError_t err;
	// // Check for kernel launch errors
	// err = hipGetLastError();
	// if (err != hipSuccess) 
 	// 	  printf("Error: %s\n", hipGetErrorString(err));
  	

  	err = hipDeviceSynchronize();
  	cout << hipGetErrorName(err) << endl;
  	
}

__global__ void magnitude(const float* outx, const float *outy, float* out, unsigned int r, unsigned int c) {
	int x_idx = threadIdx.x + (long)blockDim.x * (long)blockIdx.x;
	int y_idx = threadIdx.y + (long)blockDim.y * (long)blockIdx.y;
	int idx = y_idx*c + x_idx; // Code motion

	if (x_idx < c && y_idx < r)
		out[idx] = sqrt(outx[idx]*outx[idx] + outy[idx]*outy[idx]);
}
