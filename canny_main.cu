#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/core.hpp"
#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/imgcodecs.hpp"
#include <iostream>
#include <queue>
#include "canny.cuh"
#include "sobel.cuh"

using namespace cv;
using namespace std;

int main(int argc, char* argv[]) {
	
  	int bdx = atoi(argv[1]);
  	int bdy = atoi(argv[2]);
  	
  	hipError_t err;

  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

  	Mat image, norm_image;
    image = imread("license.jpg", 0); 	
    if(image.empty())                   
    {
        cout <<  "Could not open or find the image" << endl;
        return -1;
    }
    
    normalize(image, norm_image, 0, 1, NORM_MINMAX, CV_32F);
    float *img = norm_image.ptr<float>(0);

    // cout << image.rows << " " << image.cols << endl;

    float maskx[9] = {-1,-2,-1,0,0,0,1,2,1};
    float masky[9] = {-1,0,1,-2,0,2,-1,0,1};

    float maskx1[3] = {1, 0, -1};
    float maskx2[3] = {1, 2, 1};

    float *filter, *dimg, *supp, *outx, *outy, *output;
    
    err = hipMalloc((void **)&dimg, image.rows * image.cols * sizeof(float));
    err = hipMallocManaged((void **)&supp, image.rows * image.cols * sizeof(float));
	err = hipMallocManaged((void **)&filter, 3 * 3 * sizeof(float));
    
    float *smooth_img, *grad;
    err = hipMallocManaged((void **)&smooth_img, image.rows * image.cols * sizeof(float));
    err = hipMallocManaged((void **)&grad, image.rows * image.cols * sizeof(float));

    err = hipMemcpy(dimg, norm_image.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    
    err = hipMallocManaged((void **)&outx, image.rows * image.cols * sizeof(float));
  	err = hipMallocManaged((void **)&outy, image.rows * image.cols * sizeof(float));
  	err = hipMallocManaged((void **)&output, image.rows * image.cols * sizeof(float));
  	
  	float *dmaskx, *dmasky, *dmaskx1, *dmaskx2; // masky1 = maskx2 and masky2 = maskx1
  	err = hipMalloc((void **)&dmaskx, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmasky, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx1, 3 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx2, 3 * sizeof(float));

  	err = hipMemcpy(dmaskx, maskx, 9 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmasky, masky, 9 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmaskx1, maskx1, 3 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmaskx2, maskx2, 3 * sizeof(float), hipMemcpyHostToDevice);

  	// // Declare a queue for BFS
  	// float *queue;
  	// err = hipMalloc((void**)&queue, 2 * image.rows * image.cols * sizeof(float));
  	// int *front, *back;
  	// err = hipMallocManaged((void**)&front, sizeof(int));
  	// err = hipMallocManaged((void**)&back, sizeof(int));
  	// *front = 0, *back = 0;

	// // Lock
	// int *mutex;
	// int state = 0; // unlocked
	// hipMalloc((void **)&mutex, sizeof(int));
	// hipMemcpy(mutex, &state, sizeof(int), hipMemcpyHostToDevice);

  	int* ctr;
	hipMallocManaged((void **)&ctr, sizeof(int));
	hipMemset(ctr, 0, sizeof(int));  	

  	hipEventRecord(start);

    dim3 block(3, 3);
    dim3 grid(1, 1);
    generateGaussian<<<grid, block>>>(filter, 1.0);
    err = hipDeviceSynchronize();

    // for (int i = 0; i < 9; i++)
    // 	cout << filter[i] << " ";
    // cout << endl;
    
    
    conv(dimg, filter, smooth_img, image.rows, image.cols, bdx, bdy);
    err = hipDeviceSynchronize();
    
    Mat smoothed = Mat(image.rows, image.cols, CV_32F, smooth_img);
    Mat norm_smoothed;
    normalize(smoothed, norm_smoothed, 0, 1, NORM_MINMAX, CV_32F);
    err = hipFree(smooth_img);
    
    err = hipMemcpy(dimg, norm_smoothed.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    
    conv_opt(dimg, dmaskx1, dmaskx2, outx, image.rows, image.cols, bdx, bdy);
    conv_opt(dimg, dmaskx2, dmaskx1, outy, image.rows, image.cols, bdx, bdy);

  	block.x = bdx; block.y = bdy;
  	grid.x = (image.cols + block.x - 1) / block.x; grid.y = (image.rows + block.y - 1) / block.y;
    mag_grad<<<grid, block>>>(outx, outy, output, grad, image.rows, image.cols);
    err = hipDeviceSynchronize();
  	
    Mat mag = Mat(image.rows, image.cols, CV_32F, output);
    Mat norm_mag;
    normalize(mag, norm_mag, 0, 1, NORM_MINMAX, CV_32F);
    
	err = hipMemcpy(dimg, norm_mag.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(supp, norm_mag.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    
	NonMaxSuppression<<<grid, block>>>(grad, dimg, supp, image.rows, image.cols);
    err = hipDeviceSynchronize();
  	
	// q_init<<<grid, block>>>(supp, 0.11, queue, back, image.rows, image.cols, mutex);
	// err = hipDeviceSynchronize();

	do {

		*ctr = 0;
		hysteresis<<<grid, block>>>(supp, image.rows, image.cols, 0.08, 0.11, ctr);
		err = hipDeviceSynchronize();
		cout << *ctr << endl;
	
	} while (*ctr != 0);
	
	
	weak_disconnected_edge_removal<<<grid, block>>>(supp, image.rows, image.cols);
	err = hipDeviceSynchronize(); 
	
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);

  	Mat out = Mat(image.rows, image.cols, CV_32F, supp);
    Mat norm_out;
    normalize(out, norm_out, 0, 1, NORM_MINMAX, CV_32F);

  	cout << ms << endl;

	Mat write_out;
	normalize(norm_out, write_out, 0, 255, NORM_MINMAX, CV_8U);
	imwrite("canny1_CUDA.png", write_out);

	err = hipFree(dimg);
  	err = hipFree(filter);
    err = hipFree(grad);
  	err = hipFree(outx);
  	err = hipFree(outy);
  	err = hipFree(output);
    err = hipFree(dmaskx);
  	err = hipFree(dmasky);
  	err = hipFree(dmaskx1);
  	err = hipFree(dmaskx2);
  	err = hipFree(supp);
	// err = hipFree(mutex);
  	
  	// cout << "Done!\n";
  	
  	return 0;
}
