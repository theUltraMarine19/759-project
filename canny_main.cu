#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/core.hpp"
#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/imgcodecs.hpp"
#include <iostream>
#include <queue>
#include "canny.cuh"
#include "sobel.cuh"

using namespace cv;
using namespace std;

int main(int argc, char* argv[]) {
	
  	int bdx = atoi(argv[1]);
  	int bdy = atoi(argv[2]);
  	
  	hipError_t err;

  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

  	Mat image, norm_image;
    image = imread("license.jpg", 0); 	
    if(image.empty())                   
    {
        cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }
    
    normalize(image, norm_image, 0, 1, NORM_MINMAX, CV_32F);
    float *img = norm_image.ptr<float>(0);

    cout << image.rows << " " << image.cols << endl;

    float maskx[9] = {-1,-2,-1,0,0,0,1,2,1};
    float masky[9] = {-1,0,1,-2,0,2,-1,0,1};

    float maskx1[3] = {1, 0, -1};
    float maskx2[3] = {1, 2, 1};

    // float masky1[3] = {1, 2, 1};
    // float masky2[3] = {1, 0, -1};

    float *filter, *sigma, *dimg, *outx, *outy, *output;
    
    err = hipMalloc((void **)&dimg, image.rows * image.cols * sizeof(float));
	err = hipMallocManaged((void **)&filter, 3 * 3 * sizeof(float));
    err = hipMallocManaged((void **)&sigma, sizeof(float));

    float *smooth_img, *grad;
    err = hipMallocManaged((void **)&smooth_img, image.rows * image.cols * sizeof(float));
    err = hipMallocManaged((void **)&grad, image.rows * image.cols * sizeof(float));

    err = hipMemcpy(dimg, norm_image.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    // cout << hipGetErrorName(err) << endl;

    err = hipMallocManaged((void **)&outx, image.rows * image.cols * sizeof(float));
  	err = hipMallocManaged((void **)&outy, image.rows * image.cols * sizeof(float));
  	err = hipMallocManaged((void **)&output, image.rows * image.cols * sizeof(float));
  	
  	float *dmaskx, *dmasky, *dmaskx1, *dmaskx2; // masky1 = maskx2 and masky2 = maskx1
  	err = hipMalloc((void **)&dmaskx, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmasky, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx1, 3 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx2, 3 * sizeof(float));

  	err = hipMemcpy(dmaskx, maskx, 9 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmasky, masky, 9 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmaskx1, maskx1, 3 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmaskx2, maskx2, 3 * sizeof(float), hipMemcpyHostToDevice);
  	
  	// Can be improved with CUDA streams
  	hipEventRecord(start);

    *sigma = 1.0;
    dim3 block(3, 3);
    dim3 grid(1, 1);
    generateGaussian<<<grid, block>>>(filter, sigma);
    err = hipDeviceSynchronize();
    cout << hipGetErrorName(err) << endl;

    // for (int i = 0; i < 9; i++)
    // 	cout << filter[i] << " ";
    // cout << endl;

    conv(dimg, filter, smooth_img, image.rows, image.cols, bdx, bdy);
    err = hipDeviceSynchronize();
    // cout << hipGetErrorName(err) << endl;
    
    Mat smoothed = Mat(image.rows, image.cols, CV_32F, smooth_img);
    Mat norm_smoothed;
    normalize(smoothed, norm_smoothed, 0, 1, NORM_MINMAX, CV_32F);
    err = hipFree(smooth_img);
    // cout << hipGetErrorName(err) << endl;
    
    smooth_img = norm_smoothed.ptr<float>(0);

    err = hipMemcpy(dimg, norm_smoothed.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    cout << hipGetErrorName(err) << endl;

    conv_opt(dimg, dmaskx1, dmaskx2, outx, image.rows, image.cols, bdx, bdy);
    conv_opt(dimg, dmaskx2, dmaskx1, outy, image.rows, image.cols, bdx, bdy);

  	block.x = bdx; block.y = bdy;
  	grid.x = (image.cols + block.x - 1) / block.x; grid.y = (image.rows + block.y - 1) / block.y;
    mag_grad<<<grid, block>>>(outx, outy, output, grad, image.rows, image.cols);
    err = hipDeviceSynchronize();
  	// cout << hipGetErrorName(err) << endl;
  	
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);

  	Mat out = Mat(image.rows, image.cols, CV_32F, output);
    Mat norm_out;
    normalize(out, norm_out, 0, 1, NORM_MINMAX, CV_32F);

  	// cout << ms << endl;

	Mat write_out;
	normalize(norm_out, write_out, 0, 255, NORM_MINMAX, CV_8U);
	imwrite("canny1_CUDA.png", write_out);

	err = hipFree(dimg);
  	err = hipFree(filter);
    err = hipFree(sigma);
    err = hipFree(grad);
  	err = hipFree(outx);
  	err = hipFree(outy);
  	err = hipFree(output);
  	err = hipFree(dmaskx);
  	err = hipFree(dmasky);
  	err = hipFree(dmaskx1);
  	err = hipFree(dmaskx2);
  	
  	cout << "Done!\n";
  	
  	return 0;
}
