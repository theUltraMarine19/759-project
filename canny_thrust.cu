#include "hip/hip_runtime.h"
#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/core.hpp"
#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/imgcodecs.hpp"
#include <iostream>
#include <queue>
#include "canny.cuh"
#include "sobel.cuh"
#include <thrust/device_vector.h>
#include <thrust/transform.h>

typedef thrust::tuple<float,float> Tuple2;

using namespace cv;
using namespace std;

struct gaussian {
    int sz;
    float deno;

    gaussian(int s, float sigma) { sz = s; deno = 2 * sigma * sigma; }

    __host__ __device__ 
    float operator() (int x) {
        int y_idx = x / sz;
        int x_idx = x % sz;
        return 1.0/( exp( ( (y_idx-sz/2) * (y_idx-sz/2) + (x_idx-sz/2)*(x_idx-sz/2) )/deno ) * (deno * M_PI) );
    }
};

struct fctr {
    __host__ __device__
    float operator() (float x, float sum) {
        return x/sum;
    }
};

struct magn_grad {
    __host__ __device__
    Tuple2 operator() (float x, float y) {
        float angle = (x == 0) * 90.0 + (x != 0) * (atan2(y, x) * 180.0/M_PI);
        return Tuple2(sqrt(x*x + y*y), angle);
    }
};

int main(int argc, char* argv[]) {
	
  	int bdx = atoi(argv[2]);
  	int bdy = atoi(argv[3]);
  	
  	hipError_t err;

  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

  	Mat image, norm_image;
    image = imread(argv[1], 0); 	
    if(image.empty())                   
    {
        cout <<  "Could not open or find the image" << endl;
        return -1;
    }
    
    normalize(image, norm_image, 0, 1, NORM_MINMAX, CV_32F);
    float *img = norm_image.ptr<float>(0);

    // cout << image.rows << " " << image.cols << endl;

    float maskx[9] = {-1,-2,-1,0,0,0,1,2,1};
    float masky[9] = {-1,0,1,-2,0,2,-1,0,1};

    float maskx1[3] = {1, 0, -1};
    float maskx2[3] = {1, 2, 1};

    float *filter, *dimg, *supp, *outx, *outy, *output;
    
    err = hipMalloc((void **)&dimg, image.rows * image.cols * sizeof(float));
    err = hipMallocManaged((void **)&supp, image.rows * image.cols * sizeof(float));
	err = hipMallocManaged((void **)&filter, 3 * 3 * sizeof(float));
    
    float *smooth_img, *grad;
    err = hipMallocManaged((void **)&smooth_img, image.rows * image.cols * sizeof(float));
    err = hipMallocManaged((void **)&grad, image.rows * image.cols * sizeof(float));

    err = hipMemcpy(dimg, norm_image.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    
    err = hipMallocManaged((void **)&outx, image.rows * image.cols * sizeof(float));
  	err = hipMallocManaged((void **)&outy, image.rows * image.cols * sizeof(float));
  	err = hipMallocManaged((void **)&output, image.rows * image.cols * sizeof(float));
  	
  	float *dmaskx, *dmasky, *dmaskx1, *dmaskx2; // masky1 = maskx2 and masky2 = maskx1
  	err = hipMalloc((void **)&dmaskx, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmasky, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx1, 3 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx2, 3 * sizeof(float));

  	err = hipMemcpy(dmaskx, maskx, 9 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmasky, masky, 9 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmaskx1, maskx1, 3 * sizeof(float), hipMemcpyHostToDevice);
  	err = hipMemcpy(dmaskx2, maskx2, 3 * sizeof(float), hipMemcpyHostToDevice);

  	
  	int* ctr;
	hipMallocManaged((void **)&ctr, sizeof(int));
	hipMemset(ctr, 0, sizeof(int));  	

  	hipEventRecord(start);

    // Just need the declarations, not the values
    dim3 block(3, 3);
    dim3 grid(1, 1);

    // generateGaussian<<<grid, block>>>(filter, 1.0);
    
    thrust::device_vector<int> init(9);
    for (int i = 0; i < 9; i++)
        init[i] = i;
    gaussian op(3, 1.0);
    thrust::transform(init.begin(), init.end(), filter, op);
    float sum = thrust::reduce(filter, filter+9, 0.0, thrust::plus<float>());
    fctr op1;
    float counts[9] = {1,2,3,4,5,6,7,8,9};
    thrust::transform(filter, filter+9, counts, filter, op1);


    conv(dimg, filter, smooth_img, image.rows, image.cols, bdx, bdy);
    err = hipDeviceSynchronize();
    
    Mat smoothed = Mat(image.rows, image.cols, CV_32F, smooth_img);
    Mat norm_smoothed;
    normalize(smoothed, norm_smoothed, 0, 1, NORM_MINMAX, CV_32F);
    err = hipFree(smooth_img);
    
    err = hipMemcpy(dimg, norm_smoothed.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    
    conv_opt(dimg, dmaskx1, dmaskx2, outx, image.rows, image.cols, bdx, bdy);
    conv_opt(dimg, dmaskx2, dmaskx1, outy, image.rows, image.cols, bdx, bdy);

  	// block.x = bdx; block.y = bdy;
  	// grid.x = (image.cols + block.x - 1) / block.x; grid.y = (image.rows + block.y - 1) / block.y;
   //  mag_grad<<<grid, block>>>(outx, outy, output, grad, image.rows, image.cols);
   //  err = hipDeviceSynchronize();

    magn_grad op2;
    thrust::transform(outx, outx + image.cols*image.rows, outy, thrust::make_zip_iterator(thrust::make_tuple(output, grad)), op2);
    
    Mat mag = Mat(image.rows, image.cols, CV_32F, output);
    Mat norm_mag;
    normalize(mag, norm_mag, 0, 1, NORM_MINMAX, CV_32F);
    
	err = hipMemcpy(dimg, norm_mag.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(supp, norm_mag.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    
	NonMaxSuppression<<<grid, block, (bdx+2)*(bdy+2)*sizeof(float)>>>(grad, dimg, supp, image.rows, image.cols);
    err = hipDeviceSynchronize();
  	cout << hipGetErrorName(err) << endl;

	// q_init<<<grid, block>>>(supp, 0.11, queue, back, image.rows, image.cols, mutex);
	// err = hipDeviceSynchronize();

    // hipEventRecord(start);

    do {

		*ctr = 0;
		hysteresis<<<grid, block>>>(supp, image.rows, image.cols, 0.08, 0.11, ctr);
        // hysteresis<8, 11><<<grid, block>>>(supp, image.rows, image.cols, ctr);
		err = hipDeviceSynchronize();
		// cout << *ctr << endl;
	
	} while (*ctr != 0);
	
	
	weak_disconnected_edge_removal<<<grid, block>>>(supp, image.rows, image.cols);
	err = hipDeviceSynchronize(); 
	
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);

  	Mat out = Mat(image.rows, image.cols, CV_32F, supp);
    Mat norm_out;
    normalize(out, norm_out, 0, 1, NORM_MINMAX, CV_32F);

  	cout << ms << endl;

	Mat write_out;
	normalize(norm_out, write_out, 0, 255, NORM_MINMAX, CV_8U);
	  imwrite(argv[4], write_out);

	err = hipFree(dimg);
  	err = hipFree(filter);
    err = hipFree(grad);
  	err = hipFree(outx);
  	err = hipFree(outy);
  	err = hipFree(output);
    err = hipFree(dmaskx);
  	err = hipFree(dmasky);
  	err = hipFree(dmaskx1);
  	err = hipFree(dmaskx2);
  	err = hipFree(supp);
	// err = hipFree(mutex);
  	
  	// cout << "Done!\n";
  	
  	return 0;
}
