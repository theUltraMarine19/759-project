#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/core.hpp"
#include "/srv/home/arijit/installation/OpenCV-3.4.4/include/opencv2/imgcodecs.hpp"
#include <iostream>
#include <queue>
#include <omp.h>
#include "canny.cuh"
#include "sobel.cuh"
#include "canny.h"

using namespace cv;
using namespace std;

int main(int argc, char* argv[]) {
	
  	int bdx = atoi(argv[1]);
  	int bdy = atoi(argv[2]);
    int t = atoi(argv[3]);
  	
  	hipError_t err;

  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

  	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

    omp_set_num_threads(t);

  	Mat image, norm_image;
    image = imread("license.jpg", 0); 	
    if(image.empty())                   
    {
        cout <<  "Could not open or find the image" << endl;
        return -1;
    }
    
    normalize(image, norm_image, 0, 1, NORM_MINMAX, CV_32F);
    float *img = norm_image.ptr<float>(0);

    // cout << image.rows << " " << image.cols << endl;

    float maskx[9] = {-1,-2,-1,0,0,0,1,2,1};
    float masky[9] = {-1,0,1,-2,0,2,-1,0,1};

    float maskx1[3] = {1, 0, -1};
    float maskx2[3] = {1, 2, 1};

    float *filter, *dimg, *doutx, *douty, *doutput;
    
    err = hipMalloc((void **)&dimg, image.rows * image.cols * sizeof(float));
    err = hipMalloc((void **)&filter, 3 * 3 * sizeof(float));
	err = hipMalloc((void **)&doutx, image.rows * image.cols * sizeof(float));
  	err = hipMalloc((void **)&douty, image.rows * image.cols * sizeof(float));
  	err = hipMalloc((void **)&doutput, image.rows * image.cols * sizeof(float));
	
	float *himg, *houtx, *houty, *houtput, *hfilter;
    // err = hipHostAlloc((void **)&hfilter, 9*sizeof(float), hipHostMallocDefault);
  	err = hipHostAlloc((void **)&himg, image.rows*image.cols*sizeof(float), hipHostMallocDefault);
  	err = hipHostAlloc((void **)&houtx, image.rows*image.cols*sizeof(float), hipHostMallocDefault);
  	err = hipHostAlloc((void **)&houty, image.rows*image.cols*sizeof(float), hipHostMallocDefault);
  	err = hipHostAlloc((void **)&houtput, image.rows*image.cols*sizeof(float), hipHostMallocDefault);

    memcpy(himg, norm_image.ptr<float>(), image.rows * image.cols * sizeof(float));

    float *dgrad;
    err = hipMalloc((void **)&dgrad, image.rows * image.cols * sizeof(float));
    
    float *dmaskx, *dmasky, *dmaskx1, *dmaskx2; 			// masky1 = maskx2 and masky2 = maskx1
  	err = hipMalloc((void **)&dmaskx, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmasky, 9 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx1, 3 * sizeof(float));
  	err = hipMalloc((void **)&dmaskx2, 3 * sizeof(float));

  	float *hmaskx, *hmasky, *hmaskx1, *hmaskx2;
  	hipHostAlloc((void **)&hmaskx, 9*sizeof(float), hipHostMallocDefault);
  	hipHostAlloc((void **)&hmasky, 9*sizeof(float), hipHostMallocDefault);
  	hipHostAlloc((void **)&hmaskx1, 3*sizeof(float), hipHostMallocDefault);
  	hipHostAlloc((void **)&hmaskx2, 3*sizeof(float), hipHostMallocDefault);

  	memcpy(hmaskx, maskx, 9 * sizeof(float));
  	memcpy(hmasky, masky, 9 * sizeof(float));
  	memcpy(hmaskx1, maskx1, 3 * sizeof(float));
  	memcpy(hmaskx2, maskx2, 3 * sizeof(float));

  	
  	int* ctr;
	hipMallocManaged((void **)&ctr, sizeof(int));
	hipMemset(ctr, 0, sizeof(int));  	

	float *temp, *temp1;
	err = hipMalloc((void **)&temp, image.rows * image.cols * sizeof(float));
	err = hipMalloc((void **)&temp1, image.rows * image.cols * sizeof(float));

  	hipEventRecord(start);

    dim3 block(3, 3);
    dim3 grid(1, 1);

    err = hipMemcpyAsync(dimg, himg, image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice, stream1);
    // err = hipMemcpyAsync(filter, hfilter, 9 * sizeof(float), hipMemcpyHostToDevice, stream0);
    // generateGaussian(hfilter, 3, 1.0);

    generateGaussian<<<grid, block, 0, stream0>>>(filter, 1.0);
    
    // perform some CPU-side instructions before waiting for above 2 streams to finish
    block.x = bdx; block.y = bdy;
    grid.x = (image.cols + block.x - 1)/block.x; grid.y = (image.rows + block.y - 1)/block.y;
	err = hipDeviceSynchronize();

	conv_kernel<<<grid, block, sizeof(float) * (bdx + 2) * (bdy + 2) + 3 * 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream0>>>(dimg, filter, doutput, image.rows, image.cols);
	err = hipMemcpyAsync(dmaskx1, hmaskx1, 3 * sizeof(float), hipMemcpyHostToDevice, stream1);

	// hipStreamSynchronize(stream0); // wait for conv_kernel to finish
	// err = hipMemcpyAsync(himg, doutput, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost, stream0);

	err = hipMemcpy(himg, doutput, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost);	
    
    Mat smoothed = Mat(image.rows, image.cols, CV_32F, himg);
    Mat norm_smoothed;
    normalize(smoothed, norm_smoothed, 0, 1, NORM_MINMAX, CV_32F);
    
    // No advantage of Async since have to wait for this to finish
    err = hipMemcpy(dimg, norm_smoothed.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
    
    conv_kernel_horiz<<<grid, block, sizeof(float) * bdy * (bdx+2) + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream1>>>(dimg, dmaskx1, temp, image.rows, image.cols);
  	err = hipMemcpyAsync(dmaskx2, hmaskx2, 3 * sizeof(float), hipMemcpyHostToDevice, stream0);
  	
  	err = hipDeviceSynchronize();

  	conv_kernel_horiz<<<grid, block, sizeof(float) * bdy * (bdx+2) + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream0>>>(dimg, dmaskx2, temp1, image.rows, image.cols);
  	conv_kernel_vert<<<grid, block, sizeof(float) * (bdy+2) * bdx + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream1>>>(temp, dmaskx2, doutx, image.rows, image.cols);

  	conv_kernel_vert<<<grid, block, sizeof(float) * (bdy+2) * bdx + 3 * sizeof(float) + sizeof(float) * bdx * bdy, stream0>>>(temp1, dmaskx1, douty, image.rows, image.cols);
  	err = hipMemcpyAsync(houtx, doutx, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost, stream1);

  	mag_grad<<<grid, block, 0, stream0>>>(doutx, douty, doutput, dgrad, image.rows, image.cols);
    err = hipMemcpyAsync(houty, douty, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost, stream1);
  	
    err = hipMemcpy(houtput, doutput, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost);

    Mat mag = Mat(image.rows, image.cols, CV_32F, houtput);
    Mat norm_mag;
    normalize(mag, norm_mag, 0, 1, NORM_MINMAX, CV_32F);
    

	err = hipMemcpy(dimg, norm_mag.ptr<float>(), image.rows * image.cols * sizeof(float), hipMemcpyHostToDevice);
	err = hipMemcpy(doutput, dimg, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToDevice);
    
	NonMaxSuppression<<<grid, block, (bdx+2)*(bdy+2)*sizeof(float), stream0>>>(dgrad, dimg, doutput, image.rows, image.cols);
    
    err = hipMemcpy(houtput, doutput, image.rows * image.cols * sizeof(float), hipMemcpyDeviceToHost);

  	hysteresis(houtput, image.rows, image.cols, 0.08, 0.11);

  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);

  	Mat out = Mat(image.rows, image.cols, CV_32F, houtput);
    Mat norm_out;
    normalize(out, norm_out, 0, 1, NORM_MINMAX, CV_32F);

  	cout << ms << endl;

	Mat write_out;
	normalize(norm_out, write_out, 0, 255, NORM_MINMAX, CV_8U);
	imwrite("canny1_opt.png", write_out);

	err = hipFree(dimg);
  	err = hipFree(filter);
    err = hipFree(dgrad);
  	err = hipFree(doutx);
  	err = hipFree(douty);
  	err = hipFree(doutput);
  	err = hipFree(dmaskx);
  	err = hipFree(dmasky);
  	err = hipFree(dmaskx1);
  	err = hipFree(dmaskx2);

  	err = hipHostFree(himg);
	// err = hipHostFree(hfilter);
	err = hipHostFree(houtx);
  	err = hipHostFree(houty);
  	err = hipHostFree(houtput);
  	err = hipHostFree(hmaskx);
  	err = hipHostFree(hmasky);
  	err = hipHostFree(hmaskx1);
  	err = hipHostFree(hmaskx2);

  	return 0;
}
